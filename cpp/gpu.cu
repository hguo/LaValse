#include "rasQuery.cuh"

using namespace ras;

static Event* d_events;

__device__
bool checkTime(uint64_t t, uint64_t t0, uint64_t t1) {
  return (t >= t0 && t < t1);
}

__device__
bool checkBits(uint16_t b0, uint16_t b1) {return b0 & b1;}

__device__
bool check(uint32_t m, const bool s[]) {return s[m];}

__device__
void crossfilter_kernel(const Event& e, const Query& q, bool b[], bool c[])
{
  const int ndims = 8; // FIXME
  if (!checkTime(e.eventTime, q.T0, q.T1)) {
    for (int i=0; i<ndims; i++) {
      b[i] = false; 
      c[i] = false;
    }
    return;
  }

  b[0] = checkTime(e.eventTime, q.t0, q.t1);
  b[1] = check(e.msgID, q.msgID);
  b[2] = check(e.component(), q.component);
  b[3] = check(e.locationType, q.locationType);
  b[4] = check(e.category(), q.category);
  b[5] = check(e.severity(), q.severity);
  b[6] = check(e.location[q.LOD], q.location);
  b[7] = checkBits(e.controlActions(), q.controlActions);

  for (int i=0; i<ndims; i++) {
    bool v = true;
    for (int j=0; j<ndims; j++) {
      if (i == j) continue;
      else if (!b[j]) {
        v = false;
        break;
      }
    }
    c[i] = v;
  }
}

void uploadEvents(const std::vector<Event>& events) {
  hipMalloc((void**)&d_events, sizeof(Event)*events.size());
  hipMemcpy(d_events, events.data(), sizeof(Event)*events.size(), hipMemcpyHostToDevice);
}
